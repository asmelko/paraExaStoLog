#include "cu_context.cuh"

#include <iostream>

void cuda_check(hipError_t e)
{
	if (e != hipSuccess)
	{
		std::printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__, hipGetErrorString(e), e);
		std::exit(EXIT_FAILURE);
	}
}

void cusparse_check(hipsparseStatus_t e)
{
	if (e != HIPSPARSE_STATUS_SUCCESS)
	{
		std::printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, hipsparseGetErrorString(e), e);
		std::exit(EXIT_FAILURE);
	}
}

cu_context::cu_context() { CHECK_CUSPARSE(hipsparseCreate(&cusparse_handle)); }

cu_context::~cu_context() { CHECK_CUSPARSE(hipsparseDestroy(cusparse_handle)); }
