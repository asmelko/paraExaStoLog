#include <iostream>

#include "cu_context.h"

void cuda_check(hipError_t e, const char* file, int line)
{
	if (e != hipSuccess)
	{
		std::printf("CUDA API failed at %s:%d with error: %s (%d)\n", file, line, hipGetErrorString(e), e);
		std::exit(EXIT_FAILURE);
	}
}

void cusparse_check(hipsparseStatus_t e, const char* file, int line)
{
	if (e != HIPSPARSE_STATUS_SUCCESS)
	{
		std::printf("CUSPARSE API failed at %s:%d with error: %s (%d)\n", file, line, hipsparseGetErrorString(e), e);
		std::exit(EXIT_FAILURE);
	}
}

void cusolver_check(hipsolverStatus_t e, const char* file, int line)
{
	if (e != HIPSOLVER_STATUS_SUCCESS)
	{
		std::printf("CUSOLVER API failed at %s:%d with error: %d\n", file, line, e);
		std::exit(EXIT_FAILURE);
	}
}

cu_context::cu_context()
{
	CHECK_CUSPARSE(hipsparseCreate(&cusparse_handle));
	CHECK_CUSOLVER(hipsolverSpCreate(&cusolver_handle));
}

cu_context::~cu_context()
{
	CHECK_CUSPARSE(hipsparseDestroy(cusparse_handle));
	CHECK_CUSOLVER(hipsolverSpDestroy(cusolver_handle));
}
