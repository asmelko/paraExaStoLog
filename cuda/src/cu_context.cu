#include <iostream>

#include "cu_context.h"

void cuda_check(hipError_t e, const char* file, int line)
{
	if (e != hipSuccess)
	{
		std::printf("CUDA API failed at %s:%d with error: %s (%d)\n", file, line, hipGetErrorString(e), e);
		std::exit(EXIT_FAILURE);
	}
}

void cusparse_check(hipsparseStatus_t e, const char* file, int line)
{
	if (e != HIPSPARSE_STATUS_SUCCESS)
	{
		std::printf("CUSPARSE API failed at %s:%d with error: %s (%d)\n", file, line, hipsparseGetErrorString(e), e);
		std::exit(EXIT_FAILURE);
	}
}

void cusolver_check(hipsolverStatus_t e, const char* file, int line)
{
	if (e != HIPSOLVER_STATUS_SUCCESS)
	{
		std::printf("CUSOLVER API failed at %s:%d with error: %d\n", file, line, e);
		std::exit(EXIT_FAILURE);
	}
}

cu_context::cu_context()
{
	static bool heap_init = false;

	if (!heap_init)
	{
		size_t size;
		CHECK_CUDA(hipDeviceGetLimit(&size, hipLimitMallocHeapSize));
		std::cout << "hipLimitMallocHeapSize " << size << std::endl;
		size = 1UL << 30;
		CHECK_CUDA(hipDeviceSetLimit(hipLimitMallocHeapSize, size));
		heap_init = true;
	}
	CHECK_CUSPARSE(hipsparseCreate(&cusparse_handle));
	CHECK_CUSOLVER(hipsolverSpCreate(&cusolver_handle));
}

cu_context::~cu_context()
{
	CHECK_CUSPARSE(hipsparseDestroy(cusparse_handle));
	CHECK_CUSOLVER(hipsolverSpDestroy(cusolver_handle));
}
