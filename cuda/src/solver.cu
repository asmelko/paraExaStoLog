#include "hip/hip_runtime.h"
#include <hipsolver.h>

#include <thrust/adjacent_difference.h>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>

#include "solver.h"
#include "utils.cuh"

struct equals_ftor : public thrust::unary_function<index_t, bool>
{
	index_t value;

	equals_ftor(index_t value) : value(value) {}

	__host__ __device__ bool operator()(index_t x) const { return x == value; }
};

solver::solver(cu_context& context, const transition_table& t, transition_graph g, initial_state s)
	: context_(context),
	  initial_state_(std::move(s.state)),
	  labels_(std::move(g.labels)),
	  terminals_(std::move(g.terminals)),
	  rows_(t.rows),
	  cols_(t.cols),
	  indptr_(t.indptr)
{}

__global__ void scatter_rows_data(const __restrict__ index_t* dst_indptr, __restrict__ index_t* dst_rows,
								  __restrict__ float* dst_data, const __restrict__ index_t* src_rows,
								  const __restrict__ index_t* src_indptr, const __restrict__ index_t* src_perm,
								  int perm_size)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx >= perm_size)
		return;

	index_t src_begin = src_indptr[src_perm[idx]];
	index_t src_end = src_indptr[src_perm[idx] + 1];

	index_t dst_begin = dst_indptr[idx];

	int i = 0;
	for (; i < src_end - src_begin; i++)
	{
		dst_rows[dst_begin + i] = src_rows[src_begin + i];
	}

	dst_rows[dst_begin + i] = src_perm[idx];
	dst_data[dst_begin + i] = -(float)i;
}

float solver::determinant(const d_idxvec& indptr, const d_idxvec& rows, const thrust::device_vector<float>& data)
{
	hipsolverSpHandle_t handle;
	hipsolverSpCreate(&handle);

	thrust::host_vector<index_t> h_indptr = indptr;
	thrust::host_vector<index_t> h_rows = rows;
	thrust::host_vector<float> h_data = data;

	csrluInfoHost_t info;
	cusolverSpCreateCsrluInfoHost(&info);

	hipsparseMatDescr_t desc, descr_L, descr_U;
	CHECK_CUSPARSE(hipsparseCreateMatDescr(&desc));
	CHECK_CUSPARSE(hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
	CHECK_CUSPARSE(hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);


	CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr_L);
	CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
	CHECK_CUSPARSE(hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
	CHECK_CUSPARSE(hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
	CHECK_CUSPARSE(hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

	CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr_U);
	CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
	CHECK_CUSPARSE(hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
	CHECK_CUSPARSE(hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
	CHECK_CUSPARSE(hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

	cusolverSpXcsrluAnalysisHost(handle, indptr.size() - 1, rows.size(), desc, h_indptr.data(), h_rows.data(), info);

	size_t internal_data, workspace;
	cusolverSpScsrluBufferInfoHost(handle, indptr.size() - 1, rows.size(), desc, h_data.data(), h_indptr.data(),
								   h_rows.data(), info, &internal_data, &workspace);

	std::vector<char> buffer(workspace);

	cusolverSpScsrluFactorHost(handle, indptr.size() - 1, rows.size(), desc, h_data.data(), h_indptr.data(),
							   h_rows.data(), info, 0.f, buffer.data());

	int nnz_l, nnz_u;
	cusolverSpXcsrluNnzHost(handle, &nnz_l, &nnz_u, info);

	std::vector<index_t> P(indptr.size() - 1), Q(indptr.size() - 1), L_indptr(indptr.size()), U_indptr(indptr.size()),
		L_cols(nnz_l), U_cols(nnz_u);
	std::vector<float> L_data(nnz_l), U_data(nnz_u);

	cusolverSpScsrluExtractHost(handle, P.data(), Q.data(), descr_L, L_data.data(), L_indptr.data(), L_cols.data(),
								descr_U, U_data.data(), U_indptr.data(), U_cols.data(), info, buffer.data());

	std::vector<float> diag(indptr.size() - 1);

	thrust::for_each(thrust::host, thrust::make_counting_iterator<index_t>(0),
					 thrust::make_counting_iterator<index_t>(indptr.size() - 1), [&](index_t i) {
						 auto begin = U_indptr[i];
						 auto end = U_indptr[i + 1];

						 for (auto col_idx = begin; col_idx != end; col_idx++)
						 {
							 if (U_cols[col_idx] == i)
								 diag[i] = U_data[col_idx];
						 }
					 });

	float determinant = thrust::reduce(thrust::host, diag.begin(), diag.end(), 0, thrust::multiplies<float>());

	CHECK_CUSPARSE(hipsparseDestroyMatDescr(desc);
	CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_L);
	CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_U);
	cusolverSpDestroyCsrluInfoHost(info);
	hipsolverSpDestroy(handle);

	return determinant;
}

void solver::solve_terminal_part()
{
	// vector of vertex indices
	d_idxvec sccs(thrust::make_counting_iterator<index_t>(0), thrust::make_counting_iterator<index_t>(labels_.size()));

	// vector of terminal scc begins and ends
	std::vector<size_t> terminals_offsets;
	terminals_offsets.reserve(terminals_.size() + 1);
	terminals_offsets.push_back(0);

	d_idxvec reverse_labels(labels_.size());

	// we partition labels_ and sccs multiple times so the ordering is T1, ..., Tn, NT1, ..., NTn
	auto partition_point = thrust::make_zip_iterator(sccs.begin(), labels_.begin());
	for (auto it = terminals_.begin(); it != terminals_.end(); it++)
	{
		partition_point =
			thrust::stable_partition(partition_point, thrust::make_zip_iterator(sccs.end(), labels_.end()),
									 [terminal_idx = *it] __device__(thrust::tuple<index_t, index_t> x) {
										 return thrust::get<1>(x) == terminal_idx;
									 });

		terminals_offsets.push_back(partition_point - thrust::make_zip_iterator(sccs.begin(), labels_.begin()));
	}

	// this is point that partitions terminals and nonterminals
	auto sccs_terminals_end = partition_point;

	for (size_t i = 1; i < terminals_offsets.size(); i++)
	{
		size_t scc_size = terminals_offsets[i] - terminals_offsets[i - 1];
		d_idxvec scc_indptr(scc_size + 1);
		scc_indptr[0] = 0;

		// create map for scc vertices so they start from 0
		thrust::copy(
			thrust::make_counting_iterator<intptr_t>(0), thrust::make_counting_iterator<intptr_t>(scc_size),
			thrust::make_permutation_iterator(reverse_labels.begin(), sccs.begin() + terminals_offsets[i - 1]));

		// this creates indptr of scc in CSC
		{
			auto scc_begins_b =
				thrust::make_permutation_iterator(indptr_.begin() + 1, sccs.begin() + terminals_offsets[i - 1]);

			auto scc_begins_e =
				thrust::make_permutation_iterator(indptr_.begin() + 1, sccs.begin() + terminals_offsets[i]);

			thrust::adjacent_difference(scc_begins_b, scc_begins_e, scc_indptr.begin() + 1);

			// add 1 to each col for diagonal part
			thrust::transform(scc_indptr.begin() + 1, scc_indptr.end(), scc_indptr.begin() + 1,
							  [] __device__(index_t x) { return x + 1; });

			thrust::inclusive_scan(scc_indptr.begin(), scc_indptr.end(), scc_indptr.begin());
		}

		index_t nnz = scc_indptr.back();
		d_idxvec scc_cols(nnz), scc_rows(nnz);
		thrust::device_vector<float> scc_data(nnz, 1.f);

		// this creates rows and data of scc
		{
			int blocksize = 512;
			int gridsize = (scc_size + blocksize - 1) / blocksize;
			scatter_rows_data<<<gridsize, blocksize>>>(scc_indptr.data().get(), scc_rows.data().get(),
													   scc_data.data().get(), rows_.data().get(), indptr_.data().get(),
													   sccs.data().get() + terminals_offsets[i], scc_size);

			CHECK_CUDA(hipDeviceSynchronize());

			thrust::transform(scc_rows.begin(), scc_rows.end(), scc_rows.begin(),
							  [map = reverse_labels.data().get()] __device__(index_t x) { return map[x]; });
		}

		// this decompresses indptr into cols
		CHECK_CUSPARSE(hipsparseXcsr2coo(context_.cusparse_handle, scc_indptr.data().get(), nnz, scc_size,
										scc_cols.data().get(), HIPSPARSE_INDEX_BASE_ZERO));

		index_t row_to_remove = scc_rows.front();

		// this removes one row
		{
			auto part_point = thrust::stable_partition(thrust::make_zip_iterator(scc_rows.begin(), scc_cols.begin()),
													   thrust::make_zip_iterator(scc_rows.end(), scc_cols.end()),
													   [row_to_remove] __device__(thrust::tuple<index_t, index_t> x) {
														   return thrust::get<0>(x) != row_to_remove;
													   });

			scc_rows.resize(part_point - thrust::make_zip_iterator(scc_rows.begin(), scc_cols.begin()));
			scc_cols.resize(part_point - thrust::make_zip_iterator(scc_rows.begin(), scc_cols.begin()));
		}

		// this compresses rows back into indptr
		CHECK_CUSPARSE(hipsparseXcoo2csr(context_.cusparse_handle, scc_cols.data().get(), scc_cols.size(), scc_size,
										scc_indptr.data().get(), HIPSPARSE_INDEX_BASE_ZERO));

		// now we do minors
		d_idxvec minor_indptr(scc_size - 1), minor_rows(scc_rows.size());
		thrust::host_vector<index_t> h_scc_indptr = scc_indptr;
		for (size_t minor_i = 0; minor_i < scc_size; i++)
		{
			// copy indptr
			thrust::copy(scc_indptr.begin(), scc_indptr.begin() + i + 1, minor_indptr.begin());
			auto offset = h_scc_indptr[i + 1] - h_scc_indptr[i];
			thrust::transform(scc_indptr.begin() + i + 2, scc_indptr.end(), minor_indptr.begin() + i + 1,
							  [offset] __device__(index_t x) { x - offset; });

			// copy rows
			thrust::copy(scc_rows.begin(), scc_rows.begin() + h_scc_indptr[i], minor_rows.begin());
			thrust::copy(scc_rows.begin() + h_scc_indptr[i + 1], scc_rows.end(), minor_rows.begin() + h_scc_indptr[i]);

			// determinant
		}
	}
}

void solver::solve() {}
