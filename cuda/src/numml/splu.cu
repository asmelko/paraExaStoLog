#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <>

#include <cooperative_groups/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/set_operations.h>

#include "../solver.h"
#include "../utils.h"
#include "splu.h"

namespace cg = cooperative_groups;

template <typename T>
__device__ T* allocate(int size)
{
	auto ptr = (T*)malloc(sizeof(T) * size);
	if (ptr == NULL)
		printf("allocation failed\n");

	return ptr;
}

template <typename groupT>
__device__ index_t get_merging_data(groupT& w, const index_t* const volatile __restrict__* __restrict__ As_indices,
									const index_t* __restrict__ As_nnz, const index_t* __restrict__ work,
									index_t* __restrict__ work_indices, const index_t work_size)
{
	index_t merging_data = INT_MAX;

	for (index_t i = w.thread_rank(); i < work_size; i += w.num_threads())
	{
		const index_t row = work[i];
		index_t idx = work_indices[i];
		const index_t size = As_nnz[row];
		const volatile index_t* __restrict__ row_indices = As_indices[row];

		const index_t data = idx != size ? row_indices[idx] : INT_MAX;
		merging_data = data < merging_data ? data : merging_data;
	}

	return cg::reduce(w, merging_data, cg::less<index_t>());
}

template <typename groupT>
__device__ index_t increment_merging_data(groupT& w,
										  const index_t* const volatile __restrict__* __restrict__ As_indices,
										  const index_t* __restrict__ As_nnz, const index_t* __restrict__ work,
										  index_t* __restrict__ work_indices, const index_t work_size,
										  const index_t target)
{
	for (index_t i = w.thread_rank(); i < work_size; i += w.num_threads())
	{
		const index_t row = work[i];
		index_t idx = work_indices[i];
		const index_t size = As_nnz[row];
		const volatile index_t* row_indices = As_indices[row];

		const index_t data = idx != size ? row_indices[idx] : INT_MAX;

		idx += data == target ? 1 : 0;
		work_indices[i] = idx;
	}
}

template <typename groupT, typename indT>
__device__ index_t increment_merging_data_small(groupT& g, indT* __restrict__ row_indices, index_t& row_idx,
												const index_t row_size, const index_t target)
{
	const index_t data = row_idx != row_size ? row_indices[row_idx] : INT_MAX;
	row_idx += data == target ? 1 : 0;
}

template <typename groupT, typename indT>
__device__ index_t increment_merging_data_small(groupT& g, indT* __restrict__ row_indices, index_t& row_idx,
												const index_t row_size, const index_t curr_data, const index_t target)
{
	const bool are_same = curr_data == target;

	row_idx += are_same ? 1 : 0;

	return are_same ? (row_idx != row_size ? row_indices[row_idx] : INT_MAX) : curr_data;
}

template <typename groupT>
__device__ void set_indices(groupT& w, const index_t* const volatile __restrict__* __restrict__ As_indices,
							const index_t* __restrict__ As_nnz, const index_t* __restrict__ work,
							index_t* __restrict__ work_indices, const index_t work_size)
{
	const volatile index_t* __restrict__ As_nnz_v = As_nnz;
	for (index_t i = w.thread_rank(); i < work_size; i += w.num_threads())
	{
		const index_t row = work[i];
		index_t idx = 0;
		const index_t row_size = As_nnz_v[row];

		const volatile index_t* row_indices = As_indices[row];

		while (idx < row_size && row_indices[idx] <= row)
		{
			idx++;
		}

		work_indices[i] = idx;
	}
}

template <typename groupT>
__device__ index_t kway_merge_size(groupT& w, const index_t this_row, const index_t* __restrict__ this_row_indices,
								   const index_t this_row_size,
								   const index_t* const volatile __restrict__* __restrict__ As_indices,
								   const index_t* __restrict__ As_nnz, const index_t* __restrict__ work,
								   index_t* __restrict__ work_indices, const index_t work_size, index_t& new_work_size)
{
	if (work_size <= 31)
		return kway_merge_size_small(w, this_row, this_row_indices, this_row_size, As_indices, As_nnz, work, work_size,
									 new_work_size);
	index_t new_row_size = 0;
	new_work_size = 0;

	index_t this_row_idx = 0;

	// set indices after row
	set_indices(w, As_indices, As_nnz, work, work_indices, work_size);

	index_t l_data = get_merging_data(w, As_indices, As_nnz, work, work_indices, work_size);

	while (this_row_idx < this_row_size && l_data != INT_MAX)
	{
		index_t r_data = this_row_indices[this_row_idx];

		if (r_data == l_data)
		{
			this_row_idx++;
			increment_merging_data(w, As_indices, As_nnz, work, work_indices, work_size, l_data);
			l_data = get_merging_data(w, As_indices, As_nnz, work, work_indices, work_size);
		}
		else if (l_data < r_data)
		{
			if (l_data < this_row)
				new_work_size++;

			increment_merging_data(w, As_indices, As_nnz, work, work_indices, work_size, l_data);
			l_data = get_merging_data(w, As_indices, As_nnz, work, work_indices, work_size);
		}
		else
		{
			this_row_idx++;
		}
		new_row_size++;
	}

	// merging rows are all merged
	if (l_data == INT_MAX)
		return new_row_size + this_row_size - this_row_idx;

	if (this_row_idx == this_row_size)
	{
		while (l_data != INT_MAX)
		{
			if (l_data < this_row)
				new_work_size++;
			new_row_size++;

			increment_merging_data(w, As_indices, As_nnz, work, work_indices, work_size, l_data);
			l_data = get_merging_data(w, As_indices, As_nnz, work, work_indices, work_size);
		}
	}

	return new_row_size;
}

template <typename groupT>
__device__ index_t kway_merge_size_small(groupT& w, const index_t this_row,
										 const index_t* __restrict__ this_row_indices, const index_t this_row_size,
										 const index_t* const volatile __restrict__* __restrict__ As_indices,
										 const volatile index_t* __restrict__ As_nnz, const index_t* __restrict__ work,
										 const index_t work_size, index_t& new_work_size)
{
	if (w.thread_rank() >= work_size + 1)
		return;

	auto g = cg::coalesced_threads();

	index_t new_row_size = 0;
	new_work_size = 0;

	index_t merging_data = INT_MAX;
	index_t merging_row_idx = 0;
	const index_t merging_row = w.thread_rank() == 0 ? this_row : work[w.thread_rank() - 1];
	const index_t merging_row_size = w.thread_rank() == 0 ? this_row_size : As_nnz[merging_row];
	const volatile index_t* merging_row_indices = w.thread_rank() == 0 ? this_row_indices : As_indices[merging_row];

	// set indices after row
	if (w.thread_rank() != 0)
	{
		while (merging_row_idx < merging_row_size && merging_row_indices[merging_row_idx] <= merging_row)
		{
			merging_row_idx++;
		}
	}

	merging_data = merging_row_indices[merging_row_idx];

	index_t l_data = cg::reduce(g, merging_data, cg::less<index_t>());

	while (l_data != INT_MAX)
	{
		if (l_data != merging_data && l_data < merging_row)
			new_work_size++;
		new_row_size++;

		merging_data = increment_merging_data_small(g, merging_row_indices, merging_row_idx, merging_row_size,
													merging_data, l_data);
		l_data = cg::reduce(g, merging_data, cg::less<index_t>());
	}

	return new_row_size;
}

template <typename groupT>
__device__ void kway_merge(groupT& w, const index_t this_row, const index_t* __restrict__ this_row_indices,
						   const index_t this_row_size, const index_t* const __restrict__* __restrict__ As_indices,
						   const index_t* __restrict__ As_nnz, const index_t* __restrict__ work,
						   index_t* __restrict__ work_indices, const index_t work_size,
						   index_t* __restrict__ new_row_indices)
{
	if (work_size <= 31)
		return kway_merge_small(w, this_row, this_row_indices, this_row_size, As_indices, As_nnz, work, work_size,
								new_row_indices);

	index_t new_row_idx = 0;
	index_t this_row_idx = 0;

	// set indices after row
	set_indices(w, As_indices, As_nnz, work, work_indices, work_size);

	index_t l_data = get_merging_data(w, As_indices, As_nnz, work, work_indices, work_size);

	while (this_row_idx < this_row_size && l_data != INT_MAX)
	{
		index_t r_data = this_row_indices[this_row_idx];

		index_t to_write;

		if (r_data == l_data)
		{
			to_write = r_data;

			this_row_idx++;
			increment_merging_data(w, As_indices, As_nnz, work, work_indices, work_size, l_data);
			l_data = get_merging_data(w, As_indices, As_nnz, work, work_indices, work_size);
		}
		else if (l_data < r_data)
		{
			to_write = l_data;

			increment_merging_data(w, As_indices, As_nnz, work, work_indices, work_size, l_data);
			l_data = get_merging_data(w, As_indices, As_nnz, work, work_indices, work_size);
		}
		else
		{
			to_write = r_data;

			this_row_idx++;
		}

		if (w.thread_rank() == 0)
			new_row_indices[new_row_idx] = to_write;

		new_row_idx++;
	}

	// merging rows are all merged
	if (l_data == INT_MAX)
	{
		for (index_t i = this_row_idx + w.thread_rank(); i < this_row_size; i += w.num_threads())
			new_row_indices[new_row_idx + i - this_row_idx] = this_row_indices[i];
	}

	if (this_row_idx == this_row_size)
	{
		while (l_data != INT_MAX)
		{
			if (w.thread_rank() == 0)
				new_row_indices[new_row_idx++] = l_data;

			increment_merging_data(w, As_indices, As_nnz, work, work_indices, work_size, l_data);
			l_data = get_merging_data(w, As_indices, As_nnz, work, work_indices, work_size);
		}
	}
}

template <typename groupT>
__device__ void kway_merge_small(groupT& w, const index_t this_row, const index_t* __restrict__ this_row_indices,
								 const index_t this_row_size,
								 const index_t* const __restrict__* __restrict__ As_indices,
								 const index_t* __restrict__ As_nnz, const index_t* __restrict__ work,
								 const index_t work_size, index_t* __restrict__ new_row_indices)
{
	if (w.thread_rank() >= work_size + 1)
		return;

	auto g = cg::coalesced_threads();

	index_t new_row_idx = 0;

	index_t merging_data = INT_MAX;
	index_t merging_row_idx = 0;
	const index_t merging_row = w.thread_rank() == 0 ? this_row : work[w.thread_rank() - 1];
	const index_t merging_row_size = w.thread_rank() == 0 ? this_row_size : As_nnz[merging_row];
	const index_t* merging_row_indices = w.thread_rank() == 0 ? this_row_indices : As_indices[merging_row];

	// set indices after row
	if (w.thread_rank() != 0)
	{
		while (merging_row_idx < merging_row_size && merging_row_indices[merging_row_idx] <= merging_row)
		{
			merging_row_idx++;
		}
	}

	merging_data = merging_row_indices[merging_row_idx];

	index_t l_data = cg::reduce(g, merging_data, cg::less<index_t>());

	while (l_data != INT_MAX)
	{
		if (w.thread_rank() == 0)
			new_row_indices[new_row_idx++] = l_data;

		merging_data = increment_merging_data_small(g, merging_row_indices, merging_row_idx, merging_row_size,
													merging_data, l_data);
		l_data = cg::reduce(g, merging_data, cg::less<index_t>());
	}
}

__device__ void find_new_work(const index_t row, const index_t* __restrict__ new_indices,
							  const index_t new_indices_size, const index_t* __restrict__ old_indices,
							  const index_t old_indices_size, index_t* __restrict__ new_work)
{
	index_t new_idx = 0;
	index_t work_idx = 0;

	for (index_t old_idx = 0; old_idx < old_indices_size; old_idx++)
	{
		const index_t c = old_indices[old_idx];

		if (c > row)
			return;

		while (true)
		{
			const index_t nc = new_indices[new_idx++];

			if (nc == c)
				break;

			new_work[work_idx++] = nc;
		}
	}
}

__global__ void cuda_kernel_splu_symbolic_fact(const index_t A_rows, const index_t* __restrict__ A_indices,
											   const index_t* __restrict__ A_indptr, index_t* __restrict__ As_nnz,
											   index_t* __restrict__* __restrict__ As_indices,
											   volatile index_t* __restrict__ degree)
{
	const index_t row = (blockIdx.x * blockDim.x + threadIdx.x) / 32;

	// printf("thread %i row %i\n", blockIdx.x * blockDim.x + threadIdx.x, row);

	auto warp = cg::tiled_partition<32>(cg::this_thread_block());

	if (row >= A_rows)
		return;

	// if (warp.thread_rank() == 0)
	//	printf("row %i started\n", row);

	index_t row_size;
	index_t* row_indices;

	index_t scratchpad_alloc_size = 0;
	index_t scratchpad_size = 0;
	index_t* scratchpad = nullptr;

	{
		const index_t row_indices_begin = A_indptr[row];
		row_size = A_indptr[row + 1] - row_indices_begin;

		if (warp.thread_rank() == 0)
		{
			row_indices = allocate<index_t>(row_size);
		}

		row_indices = warp.shfl(row_indices, 0);

		for (index_t i = warp.thread_rank(); i < row_size; i += warp.num_threads())
		{
			index_t col = (A_indices + row_indices_begin)[i];
			row_indices[i] = col;
			if (col == row)
				scratchpad_alloc_size = i;
		}

		warp.sync();

		auto mask = warp.ballot(scratchpad_alloc_size);

		// if (scratchpad_alloc_size && mask == 0)
		//	printf("error\n");

		if (mask != 0)
		{
			int lane_id = -1;
			while (mask)
			{
				mask >>= 1;
				lane_id++;
			}

			scratchpad_alloc_size = warp.shfl(scratchpad_alloc_size, lane_id);
		}

		// if (bef != 0 && scratchpad_alloc_size == 0)
		//	printf("error\n");


		/*	if (scratchpad_alloc_size != 0)
				printf("thread %i scs\n", (int)warp.thread_rank());*/

		scratchpad_size = scratchpad_alloc_size;

		if (scratchpad_alloc_size)
		{
			if (warp.thread_rank() == 0)
				scratchpad = allocate<index_t>(scratchpad_size * 2);
			scratchpad = warp.shfl(scratchpad, 0);

			for (index_t i = warp.thread_rank(); i < scratchpad_size; i += warp.num_threads())
				scratchpad[i] = row_indices[i];
		}
	}

	warp.sync();

	// if (warp.thread_rank() == 0)
	//{
	//	for (index_t i = 0; i < scratchpad_size; i++)
	//		if (scratchpad[i] != row_indices[i])
	//			printf("error\n");
	// }

	index_t iteration = 0;

	while (scratchpad_size)
	{
		iteration++;

		while (true)
		{
			bool has_degree = false;
			for (index_t i = warp.thread_rank(); i < scratchpad_size; i += warp.num_threads())
			{
				const index_t index = scratchpad[i];

				has_degree |= degree[index] != 0;
			}

			has_degree = warp.any(has_degree);

			if (!has_degree)
				break;
		}

		index_t new_scratchpad_size;
		index_t new_size = kway_merge_size(warp, row, row_indices, row_size, As_indices, As_nnz, scratchpad,
										   scratchpad + scratchpad_size, scratchpad_size, new_scratchpad_size);

		new_size = warp.shfl(new_size, 0);
		new_scratchpad_size = warp.shfl(new_scratchpad_size, 0);

		if (new_size == row_size)
			break;

		// update row
		{
			index_t* row_indices_new;
			if (warp.thread_rank() == 0)
				row_indices_new = allocate<index_t>(new_size);
			row_indices_new = warp.shfl(row_indices_new, 0);

			kway_merge(warp, row, row_indices, row_size, As_indices, As_nnz, scratchpad, scratchpad + scratchpad_size,
					   scratchpad_size, row_indices_new);

			warp.sync();

			// update scratchpad
			{
				if (new_scratchpad_size > scratchpad_alloc_size)
				{
					scratchpad_alloc_size = new_scratchpad_size;

					if (warp.thread_rank() == 0)
					{
						free(scratchpad);
						scratchpad = allocate<index_t>(scratchpad_alloc_size * 2);
					}
					scratchpad = warp.shfl(scratchpad, 0);
				}

				if (warp.thread_rank() == 0)
					find_new_work(row, row_indices_new, new_size, row_indices, row_size, scratchpad);

				warp.sync();

				if (warp.thread_rank() == 0)
					printf("iteration %i row %i new work size %i old work size %i\n", iteration, row,
						   new_scratchpad_size, scratchpad_size);

				scratchpad_size = new_scratchpad_size;
			}

			/*if (warp.thread_rank() == 0)
				printf("row %i after new work \n", row);*/

			// update row size and indices
			row_size = new_size;
			if (warp.thread_rank() == 0)
				free(row_indices);
			row_indices = row_indices_new;
		}
	}

	if (warp.thread_rank() == 0)
	{
		if (scratchpad)
			free(scratchpad);

		As_nnz[row] = row_size;
		As_indices[row] = row_indices;

		__threadfence();
		degree[row] = 0;

		// printf("row %i finished\n", row);
	}
}


__global__ void cuda_kernel_splu_symbolic_populate(const index_t A_rows, const index_t* __restrict__ A_indices,
												   const index_t* __restrict__ A_indptr,
												   const real_t* __restrict__ A_data, index_t* __restrict__ As_indices,
												   const index_t* __restrict__ As_indptr, real_t* __restrict__ As_data,
												   index_t* const* const __restrict__ As_indices_by_row)
{
	const index_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= A_rows)
		return;


	const index_t out_begin = As_indptr[row];
	const index_t row_size = As_indptr[row + 1] - out_begin;
	index_t* row_indices = As_indices_by_row[row];
	As_indices += out_begin;
	As_data += out_begin;

	for (index_t i = 0; i < row_size; i++)
	{
		const index_t index = row_indices[i];
		As_indices[i] = index;
		As_data[i] = 0.f;
	}

	free(row_indices);

	const index_t orig_begin = A_indptr[row];
	const index_t orig_row_size = A_indptr[row + 1] - orig_begin;
	A_indices += orig_begin;
	A_data += orig_begin;

	index_t As_idx = 0;
	for (index_t i = 0; i < orig_row_size; i++)
	{
		const index_t v = A_indices[i];

		while (As_indices[As_idx] != v)
			As_idx++;

		// printf("row %i wrote data %f at col %i\n", row, A_data[v_i], As_idx);

		As_data[As_idx] = A_data[i];
	}
}


/**
 * Count the number of upper-triangular nonzeros for each column of a CSC matrix.
 * This is inclusive of the main diagonal.
 *
 * Indexed on columns of A.
 */
__global__ void cuda_kernel_count_U_nnz(const index_t A_rows, const index_t A_cols,
										const index_t* __restrict__ At_indices, const index_t* __restrict__ At_indptr,
										index_t* __restrict__ U_col_nnz)
{
	const index_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j >= A_cols)
		return;

	index_t nnz = 0;
	for (index_t i_i = At_indptr[j]; i_i < At_indptr[j + 1]; i_i++)
	{
		const index_t i = At_indices[i_i];
		if (i <= j)
			nnz++;
	}

	if (nnz == 0)
		printf("L has some zeros\n");

	U_col_nnz[j] = nnz;
}


/**
 * Performs a binary search on an array between i_start and i_end (inclusive).
 */
static __device__ index_t kernel_indices_binsearch(index_t i_start, index_t i_end, const index_t i_search,
												   const index_t* __restrict__ indices)
{
	index_t i_mid;
	while (i_start <= i_end)
	{
		i_mid = (i_start + i_end) / 2;
		if (indices[i_mid] < i_search)
		{
			i_start = i_mid + 1;
		}
		else if (indices[i_mid] > i_search)
		{
			i_end = i_mid - 1;
		}
		else if (indices[i_mid] == i_search)
		{
			return i_mid;
		}
	}
	return -1;
}

/**
 * The sparse numeric LU factorization from SFLU:
 * "SFLU: Synchronization-Free Sparse LU Factorization for Fast Circuit Simulation on GPUs", J. Zhao, Y. Luo, Z. Jin, Z.
 * Zhou.
 *
 * Indexed on columns of As, where As is given in CSC format and has fill-ins represented by explicit zeros.
 */
template <typename scalar_t>
__global__ void cuda_kernel_splu_numeric_sflu(const index_t A_rows, const index_t A_cols,
											  real_t* __restrict__ As_col_data,
											  const index_t* __restrict__ As_col_indices,
											  const index_t* __restrict__ As_col_indptr,
											  volatile index_t* __restrict__ degree)
{
	const index_t k = blockIdx.x * blockDim.x + threadIdx.x;
	if (k >= A_cols)
	{
		return;
	}

	index_t diag_idx;
	const index_t col_end = As_col_indptr[k + 1];
	for (index_t i_i = As_col_indptr[k]; i_i < col_end; i_i++)
	{
		const index_t i = As_col_indices[i_i];
		if (i == k)
		{
			/* Stop once we get to the diagonal. */
			diag_idx = i_i;
			break;
		}

		/* Busy wait until intermediate results are ready */
		while (degree[i] > 0)
			;

		/* Left-looking product */
		for (index_t j_i = i_i + 1; j_i < col_end; j_i++)
		{
			const index_t j = As_col_indices[j_i];
			const index_t A_ji_i =
				kernel_indices_binsearch(As_col_indptr[i], As_col_indptr[i + 1] - 1, j, As_col_indices);
			if (A_ji_i == -1)
			{
				continue;
			}
			const scalar_t A_ji = As_col_data[A_ji_i];
			const scalar_t A_ik = As_col_data[i_i];

			/* A_{jk} \gets A_{jk} - A_{ji} A_{ik} */
			As_col_data[j_i] -= A_ji * A_ik;
		}

		// printf("thread %i decremented from %i\n", k, degree[k]);
		__threadfence();
		degree[k]--;
	}

	/* Divide column of L by diagonal entry of U */
	const scalar_t A_kk = As_col_data[diag_idx];
	for (index_t i = diag_idx + 1; i < As_col_indptr[k + 1]; i++)
	{
		As_col_data[i] /= A_kk;
	}

	// printf("thread %i decremented from %i\n", k, degree[k]);
	/* Complete the factorization and update column degree */
	__threadfence();
	degree[k]--;
}

/**
 * Sparse LU Factorization, using a left-looking algorithm on the columns of A.  Based on
 * the symbolic factorization from Rose, Tarjan's fill2 and numeric factorization in SFLU.
 */
void splu(cu_context& context, const d_idxvec& A_indptr, const d_idxvec& A_indices, const d_datvec& A_data,
		  d_idxvec& As_indptr, d_idxvec& As_indices, d_datvec& As_data)
{
	const int threads_per_block = 512;

	As_indptr.resize(A_indptr.size());
	As_indptr[0] = 0;

	index_t A_rows = A_indptr.size() - 1;
	index_t A_cols = A_indptr.size() - 1;

	std::cout << "splu start " << A_rows << std::endl;

	std::cout << "splu symbolic nnz" << std::endl;

	index_t* L_nnz;
	index_t** As_indices_by_row;

	CHECK_CUDA(hipMalloc(&L_nnz, sizeof(index_t) * A_rows));
	CHECK_CUDA(hipMalloc(&As_indices_by_row, sizeof(index_t*) * A_rows));
	CHECK_CUDA(hipMemset(As_indices_by_row, 0, sizeof(index_t*) * A_rows));

	cuda_kernel_count_U_nnz<<<(A_rows + threads_per_block - 1) / threads_per_block, threads_per_block>>>(
		A_rows, A_cols, A_indices.data().get(), A_indptr.data().get(), L_nnz);

	CHECK_CUDA(hipDeviceSynchronize());
	// print("L_nnz ", d_idxvec(L_nnz, L_nnz + A_rows));

	cuda_kernel_splu_symbolic_fact<<<(A_cols + (threads_per_block / 32) - 1) / (threads_per_block / 32),
									 threads_per_block>>>(A_rows, A_indices.data().get(), A_indptr.data().get(),
														  As_indptr.data().get() + 1, As_indices_by_row, L_nnz);

	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipFree(L_nnz));

	std::cout << "splu cumsum" << std::endl;

	thrust::inclusive_scan(As_indptr.begin(), As_indptr.end(), As_indptr.begin());
	index_t As_nnz = As_indptr.back();

	As_indices.resize(As_nnz);
	As_data.resize(As_nnz);

	std::cout << "splu nnz " << As_nnz << std::endl;

	std::cout << "splu symbolic populate" << std::endl;

	cuda_kernel_splu_symbolic_populate<<<(A_cols + threads_per_block - 1) / threads_per_block, threads_per_block>>>(
		A_rows, A_indices.data().get(), A_indptr.data().get(), A_data.data().get(), As_indices.data().get(),
		As_indptr.data().get(), As_data.data().get(), As_indices_by_row);

	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipFree(As_indices_by_row));

	d_idxvec AsT_indptr, AsT_indices;
	d_datvec AsT_data;

	/* Compute the transpose/csc representation of As so that we have easy column access. */
	solver::transpose_sparse_matrix(context.cusparse_handle, As_indptr.data().get(), As_indices.data().get(),
									As_data.data().get(), A_rows, A_cols, As_data.size(), AsT_indptr, AsT_indices,
									AsT_data);
	// print("A indptr ", A_indptr);
	// print("A indice ", A_indices);
	// print("A data   ", A_data);

	// print("As indptr ", As_indptr);
	// print("As indice ", As_indices);
	// print("As data   ", As_data);

	// print("At indptr ", AsT_indptr);
	// print("At indice ", AsT_indices);
	// print("At data   ", AsT_data);

	std::cout << "splu U nnz" << std::endl;

	index_t* U_col_nnz;
	CHECK_CUDA(hipMalloc(&U_col_nnz, sizeof(index_t) * A_rows));

	/* Perform the numeric factorization on the CSC representation */
	cuda_kernel_count_U_nnz<<<(A_cols + threads_per_block - 1) / threads_per_block, threads_per_block>>>(
		A_rows, A_cols, AsT_indices.data().get(), AsT_indptr.data().get(), U_col_nnz);
	CHECK_CUDA(hipDeviceSynchronize());

	// print("splu degrees ", d_idxvec(U_col_nnz, U_col_nnz + A_cols));

	std::cout << "splu numeric" << std::endl;

	cuda_kernel_splu_numeric_sflu<real_t><<<(A_cols + threads_per_block - 1) / threads_per_block, threads_per_block>>>(
		A_rows, A_cols, AsT_data.data().get(), AsT_indices.data().get(), AsT_indptr.data().get(), U_col_nnz);

	CHECK_CUDA(hipDeviceSynchronize());
	CHECK_CUDA(hipFree(U_col_nnz));

	/* Transpose back into CSR format */
	solver::transpose_sparse_matrix(context.cusparse_handle, AsT_indptr.data().get(), AsT_indices.data().get(),
									AsT_data.data().get(), A_cols, A_rows, AsT_data.size(), As_indptr, As_indices,
									As_data);
}