#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <>

#include <cooperative_groups/reduce.h>
#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>
#include <thrust/set_operations.h>

#include "../solver.h"
#include "../sparse_utils.h"
#include "../utils.h"
#include "splu.h"

namespace cg = cooperative_groups;

constexpr size_t big_scc_threshold = 2;

__device__ index_t merge_size(const index_t this_row, const index_t* __restrict__ this_row_indices,
							  const index_t this_row_size, const index_t merging_row,
							  const index_t* __restrict__ merging_row_indices, const index_t merging_row_size)
{
	index_t this_idx = 0;
	index_t merging_idx = 0;

	index_t count = 0;

	while (merging_idx < merging_row_size && merging_row_indices[merging_idx] <= merging_row)
		merging_idx++;

	while (merging_idx < merging_row_size && this_idx < this_row_size)
	{
		const index_t this_data = this_row_indices[this_idx];
		const index_t merging_data = merging_row_indices[merging_idx];

		if (this_data == merging_data)
		{
			this_idx++;
			merging_idx++;
		}
		else if (this_data < merging_data)
		{
			this_idx++;
		}
		else
		{
			merging_idx++;
		}

		count++;
	}

	return count + this_row_size - this_idx + merging_row_size - merging_idx;
}

__device__ void merge(const index_t this_row, const index_t* __restrict__ this_row_indices,
					  const real_t* __restrict__ this_data, const index_t this_row_size, const index_t merging_row,
					  const index_t* __restrict__ merging_row_indices, const real_t* __restrict__ merging_data,
					  const index_t merging_row_size, index_t* __restrict__ out_indices, real_t* __restrict__ out_data)
{
	index_t this_idx = 0;
	index_t merging_idx = 0;

	index_t out_idx = 0;

	real_t divisor = this_data[0] / merging_data[0];
	out_data[0] = divisor;
	out_indices[0] = this_row;

	out_idx++;
	this_idx++;
	merging_idx++;


	while (merging_idx < merging_row_size && this_idx < this_row_size)
	{
		const index_t this_col = this_row_indices[this_idx];
		const index_t merging_col = merging_row_indices[merging_idx];

		if (this_col == merging_col)
		{
			out_indices[out_idx] = this_col;
			out_data[out_idx] = this_data[this_idx] - divisor * merging_data[merging_idx];
			this_idx++;
			merging_idx++;
			out_idx++;
		}
		else if (this_col < merging_col)
		{
			out_indices[out_idx] = this_col;
			out_data[out_idx] = this_data[this_idx];
			this_idx++;
			out_idx++;
		}
		else
		{
			out_indices[out_idx] = merging_col;
			out_data[out_idx] = merging_data[merging_idx];
			merging_idx++;
			out_idx++;
		}
	}

	if (merging_idx < merging_row_size)
	{
		out_indices[out_idx] = merging_row_indices[merging_idx];
		out_data[out_idx] = merging_data[merging_idx];
		merging_idx++;
		out_idx++;
	}
	else
	{
		out_indices[out_idx] = this_row_indices[merging_idx];
		out_data[out_idx] = this_data[this_idx];
		this_idx++;
		out_idx++;
	}
}



__global__ void cuda_kernel_splu_symbolic_fact_triv_nnz(const index_t sccs_rows, const index_t scc_count,
														const index_t* __restrict__ scc_sizes,
														const index_t* __restrict__ scc_offsets,
														const index_t* __restrict__ A_indices,
														const index_t* __restrict__ A_indptr,
														index_t* __restrict__ As_nnz)
{
	index_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= sccs_rows)
		return;

	const index_t scc_index = thrust::upper_bound(thrust::seq, scc_sizes, scc_sizes + scc_count, row) - scc_sizes;

	const index_t scc_offset = scc_offsets[scc_index];
	const index_t in_scc_offset = row - (scc_index == 0 ? 0 : scc_sizes[scc_index - 1]);

	const index_t scc_size = scc_index == 0 ? scc_sizes[scc_index] : scc_sizes[scc_index] - scc_sizes[scc_index - 1];

	// printf("row %i idx %i off %i inoff %i size %i\n", row, scc_index, scc_offset, in_scc_offset, scc_size);

	row = scc_offset + in_scc_offset;


	if (scc_size > big_scc_threshold)
	{
		printf("problem\n");
		return;
	}

	if (scc_size == 1 || in_scc_offset == 0)
	{
		const index_t row_indices_begin = A_indptr[row];
		index_t row_size = A_indptr[row + 1] - row_indices_begin;

		As_nnz[row] = row_size;
	}
	else
	{
		const index_t row_indices_begin = A_indptr[row];
		index_t row_size = A_indptr[row + 1] - row_indices_begin;
		const index_t* row_indices = A_indices + row_indices_begin;

		const index_t merging_row = row - 1;
		const index_t merging_row_indices_begin = A_indptr[merging_row];
		index_t merging_row_size = A_indptr[merging_row + 1] - merging_row_indices_begin;
		const index_t* merging_row_indices = A_indices + merging_row_indices_begin;

		const index_t new_row_size =
			merge_size(row, row_indices, row_size, merging_row, merging_row_indices, merging_row_size);

		As_nnz[row] = new_row_size;
	}
}

__global__ void cuda_kernel_splu_symbolic_fact_triv_populate(
	const index_t sccs_rows, const index_t scc_count, const index_t* __restrict__ scc_sizes,
	const index_t* __restrict__ scc_offsets, const index_t* __restrict__ A_indices, const real_t* __restrict__ A_data,
	const index_t* __restrict__ A_indptr, index_t* __restrict__ As_indptr, index_t* __restrict__ As_indices,
	real_t* __restrict__ As_data)
{
	index_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= sccs_rows)
		return;

	const index_t scc_index = thrust::upper_bound(thrust::seq, scc_sizes, scc_sizes + scc_count, row) - scc_sizes;

	const index_t scc_offset = scc_offsets[scc_index];
	const index_t in_scc_offset = row - (scc_index == 0 ? 0 : scc_sizes[scc_index - 1]);

	const index_t scc_size = scc_index == 0 ? scc_sizes[scc_index] : scc_sizes[scc_index] - scc_sizes[scc_index - 1];

	// printf("row %i idx %i off %i inoff %i size %i\n", row, scc_index, scc_offset, in_scc_offset, scc_size);

	row = scc_offset + in_scc_offset;

	if (scc_size > big_scc_threshold)
	{
		printf("problem\n");
		return;
	}

	if (scc_size == 1 || in_scc_offset == 0)
	{
		const index_t row_indices_begin = A_indptr[row];
		index_t row_size = A_indptr[row + 1] - row_indices_begin;

		const index_t out_row_indices_begin = As_indptr[row];

		thrust::copy(thrust::seq, A_indices + row_indices_begin, A_indices + row_indices_begin + row_size,
					 As_indices + out_row_indices_begin);

		thrust::copy(thrust::seq, A_data + row_indices_begin, A_data + row_indices_begin + row_size,
					 As_data + out_row_indices_begin);
	}
	else
	{
		const index_t row_indices_begin = A_indptr[row];
		index_t row_size = A_indptr[row + 1] - row_indices_begin;
		const index_t* row_indices = A_indices + row_indices_begin;
		const real_t* row_data = A_data + row_indices_begin;

		const index_t merging_row = row - 1;
		const index_t merging_row_indices_begin = A_indptr[merging_row];
		index_t merging_row_size = A_indptr[merging_row + 1] - merging_row_indices_begin;
		const index_t* merging_row_indices = A_indices + merging_row_indices_begin;
		const real_t* merging_row_data = A_data + merging_row_indices_begin;

		const index_t out_row_indices_begin = As_indptr[row];
		index_t* out_row_indices = As_indices + out_row_indices_begin;
		real_t* out_row_data = As_data + out_row_indices_begin;

		merge(row, row_indices, row_data, row_size, merging_row, merging_row_indices, merging_row_data,
			  merging_row_size, out_row_indices, out_row_data);
	}
}



index_t partition_sccs(const d_idxvec& scc_offsets, d_idxvec& partitioned_scc_sizes, d_idxvec& partitioned_scc_offsets)
{
	d_idxvec scc_sizes(scc_offsets.size());

	thrust::adjacent_difference(scc_offsets.begin(), scc_offsets.end(), scc_sizes.begin());

	partitioned_scc_sizes.assign(scc_sizes.begin() + 1, scc_sizes.end());
	partitioned_scc_offsets.assign(scc_offsets.begin(), scc_offsets.end() - 1);

	auto part_point = thrust::stable_partition(
		thrust::make_zip_iterator(partitioned_scc_sizes.begin(), partitioned_scc_offsets.begin()),
		thrust::make_zip_iterator(partitioned_scc_sizes.end(), partitioned_scc_offsets.end()),
		[] __device__(thrust::tuple<index_t, index_t> x) { return thrust::get<0>(x) <= big_scc_threshold; });

	index_t small_sccs = thrust::get<1>(part_point.get_iterator_tuple()) - partitioned_scc_offsets.begin();

	/*big_scc_sizes.resize(thrust::get<0>(big_scc_end.get_iterator_tuple()) - big_scc_sizes.begin());
	big_scc_offsets.resize(big_scc_sizes.size() - 1);*/

	// we need to do this because of terminals that were stored before nonterminals
	index_t base_offset = scc_offsets.front();
	thrust::transform(partitioned_scc_offsets.begin(), partitioned_scc_offsets.end(), partitioned_scc_offsets.begin(),
					  [base_offset] __device__(index_t x) { return x - base_offset; });

	thrust::inclusive_scan(partitioned_scc_sizes.begin(), partitioned_scc_sizes.begin() + small_sccs,
						   partitioned_scc_sizes.begin());
	thrust::inclusive_scan(partitioned_scc_sizes.begin() + small_sccs, partitioned_scc_sizes.end(),
						   partitioned_scc_sizes.begin() + small_sccs);

	// const index_t big_scc_rows = big_scc_sizes.back();
	// std::cout << "splu big sccs " << big_scc_sizes.size() - 1 << std::endl;
	// std::cout << "splu big scc rows " << big_scc_rows << std::endl;
	/*print("scc offs ", scc_offsets);
	print("par offs ", partitioned_scc_offsets);
	print("par size ", partitioned_scc_sizes);*/

	return small_sccs;
}

void lu_big_nnz_and_populate(const d_idxvec& scc_offsets, const d_idxvec& A_indptr, const d_idxvec& A_indices,
							 const d_datvec& A_data, d_idxvec& As_indptr)
{}

/**
 * Sparse LU Factorization, using a left-looking algorithm on the columns of A.  Based on
 * the symbolic factorization from Rose, Tarjan's fill2 and numeric factorization in SFLU.
 */
void splu(cu_context& context, const d_idxvec& scc_offsets, const d_idxvec& A_indptr, const d_idxvec& A_indices,
		  const d_datvec& A_data, d_idxvec& As_indptr, d_idxvec& As_indices, d_datvec& As_data)
{
	d_idxvec part_scc_sizes, part_scc_offsets;
	auto small_sccs_size = partition_sccs(scc_offsets, part_scc_sizes, part_scc_offsets);
	auto big_sccs_size = scc_offsets.size() - 1 - small_sccs_size;

	const index_t small_scc_rows = small_sccs_size == 0 ? 0 : part_scc_sizes[small_sccs_size - 1];
	const index_t big_scc_rows = big_sccs_size == 0 ? 0 : part_scc_sizes.back();

	std::cout << "splu big scc rows " << big_scc_rows << std::endl;
	std::cout << "splu big sccs " << big_sccs_size << std::endl;

	std::cout << "splu small scc rows " << small_scc_rows << std::endl;
	std::cout << "splu small sccs " << small_sccs_size << std::endl;

	std::cout << "splu rows " << A_indptr.size() - 1 << std::endl;

	const int threads_per_block = 512;

	As_indptr.resize(A_indptr.size());
	As_indptr[0] = 0;

	// first we count nnz of triv
	{
		cuda_kernel_splu_symbolic_fact_triv_nnz<<<(small_scc_rows + threads_per_block - 1) / threads_per_block,
												  threads_per_block>>>(
			small_scc_rows, small_sccs_size, part_scc_sizes.data().get(), part_scc_offsets.data().get(),
			A_indices.data().get(), A_indptr.data().get(), As_indptr.data().get() + 1);

		std::cout << "splu triv nnz done" << std::endl;
	}

	// without waiting we compute nnz of non triv
	{

	}

	// we allocate required space
	{
		thrust::inclusive_scan(As_indptr.begin(), As_indptr.end(), As_indptr.begin());
		index_t As_nnz = As_indptr.back();

		As_indices.resize(As_nnz);
		As_data.resize(As_nnz);
	}

	// we populate  triv
	{
		cuda_kernel_splu_symbolic_fact_triv_populate<<<(small_scc_rows + threads_per_block - 1) / threads_per_block,
													   threads_per_block>>>(
			small_scc_rows, small_sccs_size, part_scc_sizes.data().get(), part_scc_offsets.data().get(),
			A_indices.data().get(), A_indptr.data().get(), As_indptr.data().get(), As_indices.data().get(),
			As_data.data().get());

		std::cout << "splu triv populate done" << std::endl;
	}

	// we populate non triv
	{}
}