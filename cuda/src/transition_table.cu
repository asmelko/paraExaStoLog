#include <thrust/host_vector.h>
#include <thrust/set_operations.h>

#include "transition_table.h"

struct transition_ftor : public thrust::unary_function<index_t, index_t>
{
	index_t free_vars[32] = { 0 };
	index_t fixed = 0;

	transition_ftor(const std::vector<index_t>& free_v, index_t fixed)
	{
		size_t i = 0;
		for (; i < free_v.size(); i++)
			free_vars[i] = 1 << free_v[i];

		for (; i < 32; i++)
			free_vars[i] = 0;

		this->fixed = fixed;
	}
	__host__ __device__ index_t operator()(index_t x) const
	{
		index_t ret = fixed;
		for (int i = 0; i < 32; i++)
			ret += (x >> i) % 2 ? free_vars[i] : 0;

		return ret;
	}
};

d_idxvec transition_table::construct_transition_vector(const std::vector<index_t>& free_nodes, size_t fixed_val)
{
	auto c_b = thrust::make_counting_iterator(0);
	auto c_e = c_b + (1ULL << free_nodes.size());

	auto b = thrust::make_transform_iterator(c_b, transition_ftor(free_nodes, fixed_val));
	auto e = thrust::make_transform_iterator(c_e, transition_ftor(free_nodes, fixed_val));

	return d_idxvec(b, e);
}

d_idxvec generate_transitions(const std::vector<clause_t>& clauses)
{
	d_idxvec transitions;

	for (const auto& c : clauses)
	{
		auto free_vars = c.get_free_variables();
		auto fixed = c.get_fixed_part();

		d_idxvec single_clause_transitions = transition_table::construct_transition_vector(free_vars, fixed);

		d_idxvec tmp(transitions.size() + single_clause_transitions.size());

		auto tmp_end = thrust::set_union(transitions.begin(), transitions.end(), single_clause_transitions.begin(),
										 single_clause_transitions.end(), tmp.begin());

		tmp.resize(tmp_end - tmp.begin());

		std::swap(tmp, transitions);
	}

	return transitions;
}

struct flip_ftor : public thrust::unary_function<index_t, index_t>
{
	index_t mask;

	flip_ftor(index_t mask) : mask(mask) {}
	__host__ __device__ index_t operator()(index_t x) const { return x ^ mask; }
};

void transition_table::construct_table()
{
	auto p = compute_rows_and_cols();
	cols = std::move(p.first);
	rows = std::move(p.second);

	int matrix_size = (int)(1ULL << model_.nodes.size());

	size_t buffersize;
	CHECK_CUSPARSE(hipsparseXcscsort_bufferSizeExt(context_.cusparse_handle, matrix_size, matrix_size, (int)cols.size(),
												  rows.data().get(), cols.data().get(), &buffersize));

	void* d_buffer;
	hipMalloc(&d_buffer, buffersize);

	d_idxvec P(cols.size());
	CHECK_CUSPARSE(hipsparseCreateIdentityPermutation(context_.cusparse_handle, P.size(), P.data().get()));

	CHECK_CUSPARSE(hipsparseXcoosortByColumn(context_.cusparse_handle, matrix_size, matrix_size, (int)cols.size(),
											rows.data().get(), cols.data().get(), P.data().get(), d_buffer));

	indptr = d_idxvec(matrix_size + 1);

	CHECK_CUSPARSE(hipsparseXcoo2csr(context_.cusparse_handle, cols.data().get(), (int)rows.size(), matrix_size,
									indptr.data().get(), HIPSPARSE_INDEX_BASE_ZERO));
}

std::pair<d_idxvec, d_idxvec> transition_table::compute_rows_and_cols()
{
	std::vector<d_idxvec> ups, downs;

	for (const auto& f : model_.dnfs)
	{
		ups.emplace_back(generate_transitions(f.activations));
		downs.emplace_back(generate_transitions(f.deactivations));
	}

	size_t transitions_count = 0;

	for (size_t i = 0; i < ups.size(); i++)
		transitions_count += ups[i].size() + downs[i].size();

	d_idxvec trans_src(transitions_count), trans_dst(transitions_count);

	auto src_begin = trans_src.begin();
	for (size_t i = 0; i < ups.size(); i++)
	{
		src_begin = thrust::copy(ups[i].begin(), ups[i].end(), src_begin);
		src_begin = thrust::copy(downs[i].begin(), downs[i].end(), src_begin);
	}

	auto dst_begin = trans_dst.begin();
	for (size_t i = 0; i < ups.size(); i++)
	{
		dst_begin = thrust::copy(thrust::make_transform_iterator(ups[i].begin(), flip_ftor(1ULL << i)),
								 thrust::make_transform_iterator(ups[i].end(), flip_ftor(1ULL << i)), dst_begin);

		dst_begin = thrust::copy(thrust::make_transform_iterator(downs[i].begin(), flip_ftor(1ULL << i)),
								 thrust::make_transform_iterator(downs[i].end(), flip_ftor(1ULL << i)), dst_begin);
	}

	return std::make_pair(std::move(trans_src), std::move(trans_dst));
}

transition_table::transition_table(cu_context& context, model_t model) : context_(context), model_(std::move(model)) {}