#include <thrust/set_operations.h>

#include <thrust/host_vector.h>

#include "transition_table.cuh"

struct transition_ftor : public thrust::unary_function<index_t, index_t>
{
	index_t free_vars[32] = { 0 };
	index_t fixed = 0;

	transition_ftor(const std::vector<index_t>& free_v, index_t fixed)
	{
		for (size_t i = 0; i < free_v.size(); i++)
			free_vars[i] = 1ULL << free_v[i];

		this->fixed = fixed;
	}
	__host__ __device__ index_t operator()(index_t x) const
	{
		index_t ret = fixed;
		for (int i = 0; i < 32; i++)
			ret += (x >> i) % 2 ? free_vars[i] : 0;

		return ret;
	}
};

void print(const char* msg, const d_idxvec& v)
{
	thrust::host_vector<index_t> h = v;

	std::cout << msg;
	for (auto t : h)
		std::cout << t << " ";
	std::cout << std::endl;
}

d_idxvec generate_transitions(const std::vector<clause_t>& clauses)
{
	d_idxvec transitions;

	for (const auto& c : clauses)
	{
		auto free_vars = c.get_free_variables();
		auto fixed = c.get_fixed_part();

		auto c_b = thrust::make_counting_iterator(0);
		auto c_e = c_b + (1ULL << free_vars.size());

		auto b = thrust::make_transform_iterator(c_b, transition_ftor(free_vars, fixed));
		auto e = thrust::make_transform_iterator(c_e, transition_ftor(free_vars, fixed));

		d_idxvec single_clause_transitions(b, e);

		d_idxvec tmp(transitions.size() + single_clause_transitions.size());

		auto tmp_end = thrust::set_union(transitions.begin(), transitions.end(), single_clause_transitions.begin(), single_clause_transitions.end(), tmp.begin());

		tmp.resize(tmp_end - tmp.begin());

		std::swap(tmp, transitions);
	}

	return transitions;
}

struct flip_ftor : public thrust::unary_function<index_t, index_t>
{
	index_t mask;

	flip_ftor(index_t mask) : mask(mask) {}
	__host__ __device__ index_t operator()(index_t x) const { return x ^ mask; }
};

void transition_table::construct_table()
{
	auto p = compute_rows_and_cols();
	auto& trans_src = p.first;
	auto& trans_dst = p.second;

	int matrix_size = (int)(1ULL << model_.nodes.size());

	size_t buffersize;
	CHECK_CUSPARSE(hipsparseXcsrsort_bufferSizeExt(context_.cusparse_handle, matrix_size, matrix_size,
												  (int)trans_src.size(), trans_dst.data().get(), trans_src.data().get(),
												  &buffersize));

	void* d_buffer;
	hipMalloc( &d_buffer, buffersize);

	d_idxvec P(trans_src.size());
	CHECK_CUSPARSE(hipsparseCreateIdentityPermutation(context_.cusparse_handle, P.size(), P.data().get()));

	CHECK_CUSPARSE(hipsparseXcoosortByRow(context_.cusparse_handle, matrix_size, matrix_size, (int)trans_src.size(),
										 trans_dst.data().get(), trans_src.data().get(), P.data().get(), d_buffer));

	indices = std::move(trans_src);
	indptr = d_idxvec(matrix_size + 1);

	CHECK_CUSPARSE(hipsparseXcoo2csr(context_.cusparse_handle, trans_dst.data().get(), (int)trans_dst.size(), matrix_size,
									indptr.data().get(), HIPSPARSE_INDEX_BASE_ZERO));
}

std::pair<d_idxvec, d_idxvec> transition_table::compute_rows_and_cols()
{
	std::vector<d_idxvec> ups, downs;

	for (const auto& f : model_.dnfs)
	{
		ups.emplace_back(generate_transitions(f.activations));
		downs.emplace_back(generate_transitions(f.deactivations));
	}

	size_t transitions_count = 0;

	for (size_t i = 0; i < ups.size(); i++)
		transitions_count += ups[i].size() + downs[i].size();

	d_idxvec trans_src(transitions_count), trans_dst(transitions_count);

	auto src_begin = trans_src.begin();
	for (size_t i = 0; i < ups.size(); i++)
	{
		src_begin = thrust::copy(ups[i].begin(), ups[i].end(), src_begin);
		src_begin = thrust::copy(downs[i].begin(), downs[i].end(), src_begin);
	}

	auto dst_begin = trans_dst.begin();
	for (size_t i = 0; i < ups.size(); i++)
	{
		dst_begin = thrust::copy(thrust::make_transform_iterator(ups[i].begin(), flip_ftor(1ULL << i)),
								 thrust::make_transform_iterator(ups[i].end(), flip_ftor(1ULL << i)), dst_begin);

		dst_begin = thrust::copy(thrust::make_transform_iterator(downs[i].begin(), flip_ftor(1ULL << i)),
								 thrust::make_transform_iterator(downs[i].end(), flip_ftor(1ULL << i)), dst_begin);
	}

	return std::make_pair(std::move(trans_src), std::move(trans_dst));
}

transition_table::transition_table(cu_context& context, model_t model) : context_(context), model_(std::move(model)) {}